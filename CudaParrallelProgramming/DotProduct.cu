#include <stdio.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "DotProduct.h"

int main (int argc, char* argv[]) {

	//declare user input
	int rows, cols, threads, CUDA_DEVICE;
	//char *matrix, *w_vectorF;
	
	//take user arguments to parameters	
	rows = atoi(argv[1]);
	cols = atoi(argv[2]);
        //matrix = atoi(argv[3]);
        //w_vectorF = atoi(argv[4]);
	CUDA_DEVICE = atoi(argv[5]);
	threads = atoi(argv[6]);
   
	//declare pointers
        FILE *fp, *fv;

	//declare the size variable for data
	size_t data_size;
	size_t vec_Size;

	//calculate the totle size of the training data
	data_size = (size_t)((size_t)rows * (size_t)cols);

	//calculate the totol size of the weight vector data
	vec_Size = ((size_t)cols);
	

	//give space to w_vector data in cpu
	float *w_vect=(float*)malloc((vec_Size)*sizeof(float));

	//give space to train data in cpu
	float *host_train = (float*)malloc((data_size)*sizeof(float));
 
	//give space to result from device in cpu
        float* host_partition = (float*) malloc(rows*sizeof(float));

	//declare variables for spacing to GPU
        float *dev_w_vect, *dev_train, *dev_partition;
	
	//for reading data from the file to cpu
        float file_data;
	float mat[rows][cols];

	//output error if cuda device is not working
	hipError_t err = hipSetDevice(CUDA_DEVICE);
	if(err != hipSuccess) { printf("Error setting CUDA DEVICE\n"); exit(EXIT_FAILURE); }

	//make space for gpu
	hipMalloc((float**)&dev_w_vect, vec_Size*sizeof(float));
	hipMalloc((float**)&dev_train, data_size*sizeof(float));
	hipMalloc((float**)&dev_partition, rows*sizeof(float));
	
        //open train folder, output error if is not working
        fp = fopen(argv[3], "r");
	   if (fp == NULL) {
    		printf("Cannot Open the File");
		return 0;
	}

    
	//read train data in the order, map to matrix form
        int i=0;
        int j = 0;
	for(i = 0; i < rows; i++)
 	{
      		for(j = 0; j < cols; j++) 
      		{
			fscanf(fp, "%f", &file_data);
			mat[i][j] = file_data;		   
		}
	 }
	  fclose(fp);
	
	//save the data to host memory host_train
	for(int i= 0; i < cols; i++)
	  {
		for(int j = 0; j < rows; j++)
		  {   
			  host_train[rows*i+j] = mat[j][i];
		  }
	
	  }
	//open w vector file
	fv = fopen(argv[4], "r");
	if (fv == NULL) {
                printf("Cannot Open the File");
                return 0;
        }
	//read vector from w_vector file	 	  
	for(int j = 0; j < cols; j++) 
	{
		 fscanf(fv, "%f", &w_vect[j]);
	}
	  
	  fclose(fv);
		
	//host to device data transfer for both vector and traindata
	hipMemcpy(dev_w_vect, w_vect, vec_Size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_train, host_train, data_size*sizeof(float), hipMemcpyHostToDevice);
	
	//get user input thread; calculate block
	int jobs = cols;
	int BLOCKS = (jobs + threads - 1)/threads;

	//start kernel
	kernel<<<BLOCKS, threads>>>(dev_w_vect, dev_train, dev_partition, rows, cols);

	//get the result back from gpu to cpu
	hipMemcpy(host_partition,dev_partition, rows*sizeof(float), hipMemcpyDeviceToHost);
	
	//write result
	for(int i=0; i<rows; i++) {
        	printf("%f\n", host_partition[i]);
    	}
    

	hipFree(dev_w_vect);
	hipFree(dev_train);
	hipFree(dev_partition);
	free(host_partition);
}
